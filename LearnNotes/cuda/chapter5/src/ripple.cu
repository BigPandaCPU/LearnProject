#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <stdlib.h>
#include <stdio.h>
#include"../common/book.h"
#include"../common/image.h"

#define DIM 1024
#define PI 3.1415926535f

__global__ void kernel(unsigned char *ptr, int ticks)
{
	__shared__ float shared[16][16];

	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int offset = x + y * blockDim.x*gridDim.x;
	const float period = 128.0f;

	float fx = x - DIM / 2;
	float fy = y - DIM / 2;
	float d = sqrtf(fx*fx + fy * fy);
	//unsigned char grey = (unsigned char)(128.0f + 127.0f*
	//									cos(d / 10.0f - ticks / 7.0f) / 
	//									(d / 10.0f + 1.0f));
	shared[threadIdx.x][threadIdx.y] = 255 * (sinf(x*2.0f*PI / period) + 1.0f)*
		(sinf(y*2.0f*PI / period) + 1.0f) / 4.0f;
	__syncthreads();

	ptr[offset * 4 + 0] = 0;
	ptr[offset * 4 + 1] = shared[threadIdx.x][threadIdx.y];
	ptr[offset * 4 + 2] = 0;
	ptr[offset * 4 + 3] = 255;
}

struct DataBlock
{
	unsigned char *dev_bitmap;
	IMAGE *bitmap;
};

void cleanup(DataBlock* d)
{
	HANDLE_ERROR(hipFree(d->dev_bitmap));
}

int main(void)
{
	DataBlock data;
	IMAGE bitmap(DIM, DIM);
	data.bitmap = &bitmap;
	HANDLE_ERROR(hipMalloc((void**)&data.dev_bitmap, bitmap.image_size()));

	dim3 blocks(DIM / 16, DIM / 16);
	dim3 threads(16, 16);

	int ticks = 0;
	bitmap.show_image(30);
	//while (1)
	//{
	kernel <<<blocks, threads >>> (data.dev_bitmap, ticks);
	HANDLE_ERROR(hipMemcpy(data.bitmap->get_ptr(), data.dev_bitmap, data.bitmap->image_size(), hipMemcpyDeviceToHost));

	ticks++;
	bitmap.show_image();
	//if (key == 27)
	//{
	//	break;
	//}
	//}
	cleanup(&data);
	return 0;
}



